#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil.h>

unsigned int nextPow2( unsigned int x ) {
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}


////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the given reduction kernel
// For the kernels >= 3, we set threads / block to the minimum of maxThreads and
// n/2. For kernels < 3, we set to the minimum of maxThreads and n.  For kernel 
// 6, we observe the maximum specified number of blocks, because each thread in 
// that kernel can process a variable number of elements.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
	threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
	blocks = (n + (threads * 2 - 1)) / (threads * 2);
}




#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory
{
	__device__ inline operator       T*()
	{
		extern __shared__ int __smem[];
		return (T*)__smem;
	}

	__device__ inline operator const T*() const
	{
		extern __shared__ int __smem[];
		return (T*)__smem;
	}
};


template <class T, unsigned int blockSize>
__global__ void
reduce5(T *g_idata, T *g_odata, unsigned int n)
{
	// now that we are using warp-synchronous programming (below)
	// we need to declare our shared memory volatile so that the compiler
	// doesn't reorder stores to it and induce incorrect behavior.
	volatile T *sdata = SharedMemory<T>();

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;

	sdata[tid] = (i < n) ? g_idata[i]*g_idata[i] : 0;
	if (i + blockSize < n) 
		sdata[tid] += g_idata[i+blockSize] * g_idata[i+blockSize];  

	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

#ifndef __DEVICE_EMULATION__
	if (tid < 32)
#endif
	{
		if (blockSize >=  64) { sdata[tid] += sdata[tid + 32]; EMUSYNC; }
		if (blockSize >=  32) { sdata[tid] += sdata[tid + 16]; EMUSYNC; }
		if (blockSize >=  16) { sdata[tid] += sdata[tid +  8]; EMUSYNC; }
		if (blockSize >=   8) { sdata[tid] += sdata[tid +  4]; EMUSYNC; }
		if (blockSize >=   4) { sdata[tid] += sdata[tid +  2]; EMUSYNC; }
		if (blockSize >=   2) { sdata[tid] += sdata[tid +  1]; EMUSYNC; }
	}

	// write result for this block to global mem 
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


float GPUPowSum(float* gpu_diff, int length)
{
	float sum = 0;
	int maxBlocks = 64;
	int maxThreads = 256;
	int numThreads = 0;
	int numBlocks = 0;
	getNumBlocksAndThreads(length, maxBlocks, maxThreads, numBlocks, numThreads);


	dim3 dimBlock(numThreads, 1, 1);
	dim3 dimGrid(numBlocks, 1, 1);
	int smemSize = (numThreads <= 32) ? 2 * numThreads * sizeof(float) : numThreads * sizeof(float);


	float* d_odata;
	float* h_odata;
	h_odata = (float*) malloc(sizeof(float) * numBlocks);

	CUDA_SAFE_CALL(hipMalloc((void**) &d_odata, sizeof(float)* numBlocks));


	switch (numThreads)
	{
	case 512:
		reduce5<float, 512><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	case 256:
		reduce5<float, 256><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	case 128:
		reduce5<float, 128><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	case 64:
		reduce5<float,  64><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	case 32:
		reduce5<float,  32><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	case 16:
		reduce5<float,  16><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	case  8:
		reduce5<float,   8><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	case  4:
		reduce5<float,   4><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	case  2:
		reduce5<float,   2><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	case  1:
		reduce5<float,   1><<< dimGrid, dimBlock, smemSize >>>(gpu_diff, d_odata, length); break;
	}

	CUDA_SAFE_CALL(hipMemcpy(h_odata, d_odata, sizeof(float)* numBlocks, hipMemcpyDeviceToHost));

	for(int i=0;i<numBlocks;i++)
	{
		//printf("block [%d] = %.6f\n", i, h_odata[i]);
		sum += h_odata[i];
	}


	CUDA_SAFE_CALL(hipFree(d_odata));
	free(h_odata);

	return sum;

}

float HostPowSum(float* diff, int length)
{
	float sum = 0;

	float* d_diff;
	
	
	CUDA_SAFE_CALL(hipMalloc((void**) &d_diff, sizeof(float)* length));
	
	CUDA_SAFE_CALL(hipMemcpy(d_diff, diff, length*sizeof(float), hipMemcpyHostToDevice));

	sum = GPUPowSum(d_diff, length);
	
	CUDA_SAFE_CALL(hipFree(d_diff));
	
	return sum;

}

