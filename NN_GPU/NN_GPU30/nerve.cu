#include "hip/hip_runtime.h"
/*
* Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.  Users and possessors of this source code
* are hereby granted a nonexclusive, royalty-free license to use this code
* in individual and commercial software.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.   This source code is a "commercial item" as
* that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer  software"  and "commercial computer software
* documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*
* Any use of this source code in individual and commercial software must
* include, in the user documentation and internal comments to the code,
* the above Disclaimer and U.S. Government End Users Notice.
*/

/* BP nerve neworl research
* Host code.
*/
#include "nerve.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil.h>
#include <hipblas.h>
#include <nerve_kernel.h>
#include "Image.h"
#include "Reduce.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

////////////////////////////////////////////////////////////////////////////////
// declaration, forward




////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
const int OUTPUT_NUM = 4;

const int TARGET_NUM = 10;
float target[TARGET_NUM][OUTPUT_NUM];

const float HIGH = 1.0f;
const float LOW = 0.0f;

int
run(int argc, char** argv)
{	
	const float lr = 0.1f;
	const int MaxEpochs = 1;
	const int HiddenUnitNum = 10;
	const int InDim = 2;
	const int OutDim = 3;
	const int SamNum = 200;

	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));

	float* h_SamInEx;
	float* h_SamOut;
	float* h_W1Ex;
	float* h_W2Ex;
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	CUT_DEVICE_INIT(argc, argv);
	printf("%s\n", hipGetErrorString(hipGetLastError()));


	hipblasStatus_t status;
	status = cublasInit();
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		printf("Can't init cublas\n");
		printf("%s\n", hipGetErrorString(hipGetLastError()));
		return -1;
	}

	printf("%s\n", hipGetErrorString(hipGetLastError()));


	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamInEx, sizeof(float)*size_SamInEx));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamOut, sizeof(float)*size_SamOut));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_W1Ex, sizeof(float)*size_W1Ex));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_W2Ex, sizeof(float)*size_W2Ex));

	if(!InitSample(SamNum, InDim, OutDim, HiddenUnitNum, h_SamInEx, h_SamOut, h_W1Ex, h_W2Ex))
	{
		printf("Can't init sample\n");
		return 0;
	}
	train(lr, MaxEpochs, HiddenUnitNum, InDim, OutDim, SamNum, h_SamInEx, h_SamOut, h_W1Ex, h_W2Ex, false, 0.0f);
	/*
	printf("\n");
	for(int i=0; i<size_W1Ex; i++)
	{
	printf("%3.3f ",h_W1Ex[i]);
	}

	printf("\n");
	for(int i=0; i<size_W2Ex; i++)
	{
	printf("%3.3f ",h_W2Ex[i]);
	}
	*/
	FILE *p;
	p = fopen("nervedata0_W1Ex.dat", "wb");
	fwrite(h_W1Ex,sizeof(float),size_W1Ex,p);
	fclose(p);

	p = fopen("nervedata0_W2Ex.dat", "wb");
	fwrite(h_W2Ex,sizeof(float),size_W2Ex,p);
	fclose(p);


	CUDA_SAFE_CALL( hipHostFree((h_SamInEx)));
	CUDA_SAFE_CALL( hipHostFree((h_SamOut)));
	CUDA_SAFE_CALL( hipHostFree((h_W1Ex)));
	CUDA_SAFE_CALL( hipHostFree((h_W2Ex)));

	h_SamInEx = NULL;
	h_SamOut = NULL;
	h_W1Ex=NULL;
	h_W2Ex=NULL;

	cublasShutdown();
	CUT_EXIT(argc, argv);

	printf("run complete\n");
}


int runImage(int argc, char** argv, Image* imageList, int trainnum, int testnum, int maxIter, bool changeEta, float maxtime)
{
	if(imageList == NULL || trainnum == 0)
	{
		return -1;
	}
	
	const float lr = 0.1f;
	const int MaxEpochs = maxIter;
	const int HiddenUnitNum = 16;
	const int InDim = imageList[0].length;
	const int OutDim = OUTPUT_NUM;
	int SamNum = trainnum;

	int size_SamInEx = ((int)(SamNum*(InDim+1)));
	int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));

	float* h_SamInEx;
	float* h_SamOut;
	float* h_W1Ex;
	float* h_W2Ex;
	
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamInEx, sizeof(float)*size_SamInEx));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamOut, sizeof(float)*size_SamOut));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_W1Ex, sizeof(float)*size_W1Ex));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_W2Ex, sizeof(float)*size_W2Ex));
	
	if(!InitImage(SamNum, InDim, OutDim, HiddenUnitNum, h_SamInEx, h_SamOut, h_W1Ex, h_W2Ex, imageList))
	{
		printf("Can't init image into input\n");
		return 0;
	}
	train(lr, MaxEpochs, HiddenUnitNum, InDim, OutDim, SamNum, h_SamInEx, h_SamOut, h_W1Ex, h_W2Ex, changeEta, maxtime);

	if(testnum != 0)
	{
		if(testnum != trainnum)
		{
			SamNum = testnum;
			size_SamInEx = ((int)(SamNum*(InDim+1)));
			size_SamOut =  ((int)(SamNum*OutDim));
			CUDA_SAFE_CALL( hipHostFree((h_SamOut)));
			CUDA_SAFE_CALL( hipHostFree((h_SamInEx)));
			CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamInEx, sizeof(float)*size_SamInEx));
			CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamOut, sizeof(float)*size_SamOut));
		}
		
		Image* testEntrance = imageList+trainnum;
		for(int i=0; i<SamNum; i++)
		{	
			for(int j=0; j<InDim;j++)
			{
				h_SamInEx[IDX2C(i,j,SamNum)] = testEntrance[i].content[j] / 16;
			}
		}
		Test(h_SamInEx, InDim, HiddenUnitNum, OutDim, SamNum, h_W1Ex, h_W2Ex, testEntrance);
	}
	

	CUDA_SAFE_CALL( hipHostFree((h_SamInEx)));
	CUDA_SAFE_CALL( hipHostFree((h_SamOut)));
	CUDA_SAFE_CALL( hipHostFree((h_W1Ex)));
	CUDA_SAFE_CALL( hipHostFree((h_W2Ex)));

	h_SamInEx = NULL;
	h_SamOut = NULL;
	h_W1Ex=NULL;
	h_W2Ex=NULL;

	

	printf("run complete\n");
}

int 
iDivUp(int a, int b){
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}


bool InitImage(int SamNum, int InDim, int OutDim, int HiddenUnitNum, float* h_SamInEx, float* h_SamOut, float* h_W1Ex,float* h_W2Ex, Image* imageList)
{
	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)((InDim+1)*HiddenUnitNum));
	const int size_W2Ex	 = ((int)((HiddenUnitNum+1)*OutDim));
	


	for(int i=0; i<size_SamInEx; i++)
	{
		h_SamInEx[i] = 1.0f;
	}

	for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<InDim;j++)
		{
			h_SamInEx[IDX2C(i,j,SamNum)] = imageList[i].content[j] / 16;
		}
	}
	/*
	#ifdef _DEBUG
	printf("Input:\n");
	for(int i=0; i<SamNum; i++)
	{	
	for(int j=0; j<InDim;j++)
	{
	printf("%.6f\t", h_SamInEx[IDX2C(i,j,SamNum)]);
	}
	printf("\n");
	}
	#endif // _DEBUG
	*/

	
	for(int i=0;i<TARGET_NUM;i++)
	{
		int fi = i;
		int j = OUTPUT_NUM -1;
		memset(target[i], 0, sizeof(float) * OUTPUT_NUM);
		while(fi)
		{
			target[i][j--] =(float)( fi & 1 ? HIGH : LOW);
			fi >>= 1;
		}
	}

	for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<OutDim;j++)
		{

			h_SamOut[IDX2C(i,j,SamNum)] = target[imageList[i].label][j];
			//printf("%.2f\t", h_SamOut[IDX2C(i,j,SamNum)]);
		}
		//printf("\n");
	}
	/*
	#ifdef _DEBUG
	printf("Output:\n");
	for(int i=0; i<SamNum; i++)
	{	
	for(int j=0; j<OutDim;j++)
	{

	printf("%.3f\t",h_SamOut[IDX2C(i,j,SamNum)]);
	}
	printf("\n");
	}
	#endif // _DEBUG
	*/
	

	FILE* fptr = fopen("w1.txt", "r");
	FILE* fptr2 = fopen("w2.txt", "r");
	if(fptr != NULL && fptr2 != NULL)
	{
		printf("Weight files exist, reading files...\n");
		for(int i=0; i<InDim+1; i++)
		{
			for(int j=0;j<HiddenUnitNum;j++)
			{
				fscanf(fptr, "%f", &h_W1Ex[IDX2C(i,j,InDim+1)]);
			}

		}
		

		for(int i=0; i<HiddenUnitNum+1; i++)
		{
			for(int j=0;j<OutDim;j++)
			{
				fscanf(fptr2, "%f", &h_W2Ex[IDX2C(i,j,HiddenUnitNum+1)]);
				
			}

		}

		fclose(fptr);
		fclose(fptr2);
	}
	else
	{
		srand(time(NULL));
		printf("Files not found completely, generating...\n");
		for(int i=0; i<size_W1Ex; i++)
		{
			h_W1Ex[i]=0.2f*rand()/(float)RAND_MAX - 0.1f;
		}
		for(int i=0; i<size_W2Ex; i++)
		{
			h_W2Ex[i]=0.2f*rand()/(float)RAND_MAX - 0.1f;
		}

		freopen("w1.txt", "w", stdout);
		for(int i=0; i<InDim+1; i++)
		{
			for(int j=0;j<HiddenUnitNum;j++)
			{
				printf("%f\t",h_W1Ex[IDX2C(i,j,InDim+1)]);
			}

		}

		freopen("w2.txt", "w", stdout);
		for(int i=0; i<HiddenUnitNum+1; i++)
		{
			for(int j=0;j<OutDim;j++)
			{
				printf("%f\t", h_W2Ex[IDX2C(i,j,HiddenUnitNum)]);
			}

		}

		freopen("CON", "w",stdout);
	}
	

	
	
	
	
	
	return true;
}

bool InitSample(int SamNum, int InDim, int OutDim, int HiddenUnitNum, float* h_SamInEx, float* h_SamOut, float* h_W1Ex,float* h_W2Ex)
{
	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));


	srand(clock());
	FILE *p;
	p = fopen("nervedata\\samin.dat", "rb");

	if(!p) return false;

	for(int i=0; i<size_SamInEx; i++)
	{
		h_SamInEx[i] = 1.0f;
	}

	for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<InDim;j++)
		{
			fread(&h_SamInEx[j*SamNum+i],sizeof(float),1,p);
		}
	}
	fclose(p);

	for(int i=0; i<size_SamInEx; i++)
	{
		//printf("%3.4f\t",h_SamInEx[i]);
		//if((i + 1) % InDim == 0) printf("\n");
	}

	printf("\n");

	p = fopen("nervedata\\samout.dat", "rb");

	if(!p) return false;

	for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<OutDim;j++)
		{
			fread(&h_SamOut[j*SamNum+i],sizeof(float),1,p);
		}
	}
	for(int i=0; i<size_SamOut; i++)
	{
		//printf("%3.5f\t",h_SamOut[i]);
		//if((i + 1) % OutDim == 0) printf("\n");
	}
	fclose(p);

	printf("\n");

	for(int i=0; i<size_W1Ex; i++)
	{
		h_W1Ex[i]=0.2f*rand()/(float)RAND_MAX - 0.1f;
	}
	for(int i=0; i<size_W2Ex; i++)
	{
		h_W2Ex[i]=0.2f*rand()/(float)RAND_MAX - 0.1f;
	}
	/*
	p = fopen("I:\\temp\\NN_GPU\\nervedataW1Ex.dat", "rb");
	for(int i=0; i<(InDim+1); i++)
	{	
	for(int j=0; j<HiddenUnitNum;j++)
	{
	fread(&h_W1Ex[j*(InDim+1)+i],sizeof(float),1,p);
	}
	}

	for(int i=0; i<size_W1Ex; i++)
	{
	printf("%3.3f ",h_W1Ex[i]);
	}
	fclose(p);

	printf("\n");

	p = fopen("I:\\temp\\NN_GPU\\nervedataW2Ex.dat", "rb");
	for(int i=0; i<(HiddenUnitNum+1); i++)
	{	
	for(int j=0; j<OutDim;j++)
	{
	fread(&h_W2Ex[j*(HiddenUnitNum+1)+i],sizeof(float),1,p);
	}
	}

	for(int i=0; i<size_W2Ex; i++)
	{
	printf("%3.3f ",h_W2Ex[i]);
	}
	fclose(p);
	*/
	printf("\n");

	p=NULL;

	return true;
}



void
train(float lr, int MaxEpochs, int HiddenUnitNum, int InDim,int OutDim,int SamNum, float* h_SamInEx, float* h_SamOut, float* h_W1Ex, float* h_W2Ex, bool changeEta, float maxtime)
{
	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));

	float* d_W1Ex;
	float* d_W2Ex;
	float* d_W2;
	float* d_SamInEx;
	float* d_SamOut;
	float* HiddenOutEx;
	float* NetworkOut;
	float* Delta1;
	float* Delta2;
	float alpha = 1.05;
	float beta = 0.7;

	float preCopyTime, runTime, postCopyTime;
	

	unsigned int timer = 0;
	unsigned int copyTimer = 0;
	unsigned int postCopyTimer = 0;

/*
	for(int i=0;i<SamNum;i++)
	{
		for(int j=0;j<InDim+1;j++)
		{
			printf("%.4f\t", h_SamInEx[IDX2C(i,j,SamNum)]);
		}
		printf("\n");
	}
*/
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 HiddenOutExgrid(iDivUp(SamNum, BLOCK_SIZE), iDivUp((HiddenUnitNum+1), BLOCK_SIZE));
	dim3 HiddenOutgrid(iDivUp(SamNum, BLOCK_SIZE), iDivUp((HiddenUnitNum), BLOCK_SIZE));
	dim3 SamOutgrid(iDivUp(SamNum, BLOCK_SIZE), iDivUp(OutDim, BLOCK_SIZE));

	dim3 OutErrGrid(iDivUp(OutDim, BLOCK_SIZE), iDivUp(OutDim, BLOCK_SIZE));

	float* h_HiddenOutEx;
	h_HiddenOutEx = (float*) malloc(sizeof(float)*(HiddenUnitNum+1)*SamNum);
	CUT_SAFE_CALL( cutCreateTimer( &copyTimer));
	CUT_SAFE_CALL( cutStartTimer( copyTimer));
	

	CUDA_SAFE_CALL(hipMalloc((void**) &d_W1Ex, sizeof(float)*size_W1Ex));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_W2Ex, sizeof(float)*size_W2Ex));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_W2, sizeof(float)*OutDim*HiddenUnitNum));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_SamInEx, sizeof(float)*size_SamInEx));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_SamOut, sizeof(float)*size_SamOut));
	CUDA_SAFE_CALL(hipMalloc((void**) &HiddenOutEx, sizeof(float)*(HiddenUnitNum+1)*SamNum));
	CUDA_SAFE_CALL(hipMalloc((void**) &NetworkOut, sizeof(float)*size_SamOut));
	CUDA_SAFE_CALL(hipMalloc((void**) &Delta1, sizeof(float)*HiddenUnitNum*SamNum));
	CUDA_SAFE_CALL(hipMalloc((void**) &Delta2, sizeof(float)*size_SamOut));  
	CUDA_SAFE_CALL(hipMemcpy(d_W1Ex, h_W1Ex, sizeof(float)*size_W1Ex,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_W2Ex, h_W2Ex, sizeof(float)*size_W2Ex,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_SamInEx, h_SamInEx, sizeof(float)*size_SamInEx,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_SamOut, h_SamOut, sizeof(float)*size_SamOut,hipMemcpyHostToDevice));
	
	
	CUT_SAFE_CALL( cutStopTimer( copyTimer));
	preCopyTime = cutGetTimerValue( copyTimer);
	printf( "Pre-copy time: %f (ms)\n", cutGetTimerValue( copyTimer));
	CUT_SAFE_CALL( cutDeleteTimer( copyTimer));
	
	CUT_SAFE_CALL( cutCreateTimer( &timer));
	CUT_SAFE_CALL( cutStartTimer( timer));
	float errOld;
	float errNew;
	int forwardCount = 0;

	forwardCount++;
	for(int i=0; i< OutDim; i++)
	{
		CUDA_SAFE_CALL(hipMemcpy(d_W2 + HiddenUnitNum * i, d_W2Ex + (HiddenUnitNum + 1) * i, sizeof(float)*HiddenUnitNum,hipMemcpyDeviceToDevice));
		//ע��������column-majoring
	}
	hipblasSgemm('n','n',SamNum, HiddenUnitNum, (InDim+1), 1.0f,  d_SamInEx, SamNum, d_W1Ex, (InDim+1), 0.0f, HiddenOutEx, SamNum );
	

	

	logsig1<<<HiddenOutExgrid, threads>>>(HiddenOutEx, SamNum, HiddenUnitNum);

	//logsig1��߽����ֵ��Ϊ1
	hipblasSgemm('n','n',SamNum, OutDim, (HiddenUnitNum+1), 1.0f,  HiddenOutEx, SamNum, d_W2Ex, (HiddenUnitNum+1), 0.0f, NetworkOut, SamNum );
	logsig2<<<SamOutgrid,threads>>>(NetworkOut, SamNum, OutDim);
	//logsig2����߽���ֵ
	
	
	
	

	dotsub<<<SamOutgrid,threads>>>(Delta2, d_SamOut, NetworkOut, SamNum, OutDim);
	//#ifdef VERBOSE
	errOld = GPUPowSum(Delta2, OutDim* SamNum);
	for(int l=0; l< MaxEpochs; l++)
	{



		getdelta<<<SamOutgrid,threads>>>(Delta2, NetworkOut, SamNum, OutDim);
		
		hipblasSgemm('t','n',(HiddenUnitNum+1), OutDim, SamNum, lr,  HiddenOutEx, SamNum, Delta2, SamNum, 1.0f, d_W2Ex, (HiddenUnitNum+1) );

		//ʵ���Ͻ�W�仯����������
		hipblasSgemm('n','t', SamNum, HiddenUnitNum, OutDim, 1.0f,  Delta2, SamNum, d_W2, HiddenUnitNum, 0.0f, Delta1, SamNum );
		
		getdelta<<<HiddenOutgrid, threads>>>(Delta1, HiddenOutEx, SamNum, HiddenUnitNum);
		
		
		hipblasSgemm('t','n', (InDim+1), HiddenUnitNum, SamNum, lr, d_SamInEx, SamNum, Delta1, SamNum, 1.0f, d_W1Ex, (InDim+1));
		

		forwardCount++;
		for(int i=0; i< OutDim; i++)
		{
			CUDA_SAFE_CALL(hipMemcpy(d_W2 + HiddenUnitNum * i, d_W2Ex + (HiddenUnitNum + 1) * i, sizeof(float)*HiddenUnitNum,hipMemcpyDeviceToDevice));
			//ע��������column-majoring
		}

		hipblasSgemm('n','n',SamNum, HiddenUnitNum, (InDim+1), 1.0f,  d_SamInEx, SamNum, d_W1Ex, (InDim+1), 0.0f, HiddenOutEx, SamNum );
		//SamNum*(InDim+1) * (InDim+1)*HiddenUnitNUm
		
		#ifdef _DEBUG
		CUDA_SAFE_CALL(hipMemcpy(h_HiddenOutEx, HiddenOutEx, sizeof(float)*(HiddenUnitNum+1)*SamNum, hipMemcpyDeviceToHost));
		#endif // _DEBUG
		

		logsig1<<<HiddenOutExgrid, threads>>>(HiddenOutEx, SamNum, HiddenUnitNum);
		//logsig1��߽����ֵ��Ϊ1
		hipblasSgemm('n','n',SamNum, OutDim, (HiddenUnitNum+1), 1.0f,  HiddenOutEx, SamNum, d_W2Ex, (HiddenUnitNum+1), 0.0f, NetworkOut, SamNum );
		logsig2<<<SamOutgrid,threads>>>(NetworkOut, SamNum, OutDim);
		//logsig2����߽���ֵ
		dotsub<<<SamOutgrid,threads>>>(Delta2, d_SamOut, NetworkOut, SamNum, OutDim);

		errNew = GPUPowSum(Delta2, OutDim* SamNum);
	
		#ifdef _DEBUG
			printf("Iter = %d, New = %.6f, Old = %.6f, Eta = %.3f\n", l, errNew, errOld, lr);
		#endif // _DEBUG
		if(errNew < errOld)
		{
			errOld = errNew;
			if(changeEta)
			{
				float newEta = lr * 1.05;
				lr = newEta < 0.9 ? newEta : lr ;
			}
		}
		else if(errNew > errOld * 1.04)
		{
			if(changeEta)
			{
				float newEta = lr* 0.7;
				lr = newEta > 0.01 ? newEta : lr;
			}
		}
	
		if(maxtime != 0)
		{
			cutStopTimer(timer);
			float nowtime =  cutGetTimerValue( timer);
			cutStartTimer(timer);
			if(nowtime > maxtime)
			{
				
				break;
			}
		}

		

	}
	CUT_SAFE_CALL( cutStopTimer( timer));
	runTime = cutGetTimerValue( timer);
	printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
	CUT_SAFE_CALL( cutDeleteTimer( timer));
	printf("Err = %.6f\n",errNew);
	printf("Forward Count = %d\n", forwardCount);

	CUT_SAFE_CALL( cutCreateTimer( &postCopyTimer));
	CUT_SAFE_CALL( cutStartTimer( postCopyTimer));
	CUDA_SAFE_CALL(hipMemcpy(h_W1Ex, d_W1Ex, sizeof(float)*size_W1Ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(h_W2Ex, d_W2Ex, sizeof(float)*size_W2Ex,hipMemcpyDeviceToHost));

	CUT_SAFE_CALL( cutStopTimer( postCopyTimer));
	postCopyTime = cutGetTimerValue( postCopyTimer);
	printf( "Post-copy time: %f (ms)\n", cutGetTimerValue( postCopyTimer));
	CUT_SAFE_CALL( cutDeleteTimer( postCopyTimer));


	printf("Copy time percent = %.4f %%\n", (preCopyTime + postCopyTime) * 100 / (preCopyTime + postCopyTime + runTime) );

	
	#ifdef _DEBUG
	for(int i=0;i<SamNum;i++)
	{
	for(int j = 0;j<HiddenUnitNum +1;j++)
	{
	printf("%.3f\t", h_HiddenOutEx[IDX2C(i,j,SamNum)]);
	}
	printf("\n");
	}
	#endif // _DEBUG
	

	CUDA_SAFE_CALL(hipFree(d_SamInEx));
	CUDA_SAFE_CALL(hipFree(d_SamOut));
	CUDA_SAFE_CALL(hipFree(d_W1Ex));
	CUDA_SAFE_CALL(hipFree(d_W2Ex));
	CUDA_SAFE_CALL(hipFree(d_W2));

	CUDA_SAFE_CALL(hipFree(HiddenOutEx));
	CUDA_SAFE_CALL(hipFree(NetworkOut));
	CUDA_SAFE_CALL(hipFree(Delta1));
	CUDA_SAFE_CALL(hipFree(Delta2));

	free(h_HiddenOutEx);
}

float Test(float* h_SamInEx, int InDim, int HiddenUnitNum, int OutDim ,int SamNum, float* h_W1Ex, float* h_W2Ex, Image* imageList)
{
	printf("Begin to test...\n");
	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));

	float* d_W1Ex;
	float* d_W2Ex;
	float* d_W2;
	float* d_SamInEx;
	float* HiddenOutEx;
	float* NetworkOut;


	CUDA_SAFE_CALL(hipMalloc((void**) &d_W1Ex, sizeof(float)*size_W1Ex));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_W2Ex, sizeof(float)*size_W2Ex));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_W2, sizeof(float)*OutDim*HiddenUnitNum));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_SamInEx, sizeof(float)*size_SamInEx));
	CUDA_SAFE_CALL(hipMalloc((void**) &HiddenOutEx, sizeof(float)*(HiddenUnitNum+1)*SamNum));
	CUDA_SAFE_CALL(hipMalloc((void**) &NetworkOut, sizeof(float)*size_SamOut));

	CUDA_SAFE_CALL(hipMemcpy(d_W1Ex, h_W1Ex, sizeof(float)*size_W1Ex,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_W2Ex, h_W2Ex, sizeof(float)*size_W2Ex,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_SamInEx, h_SamInEx, sizeof(float)*size_SamInEx,hipMemcpyHostToDevice));

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 HiddenOutExgrid(iDivUp(SamNum, BLOCK_SIZE), iDivUp((HiddenUnitNum+1), BLOCK_SIZE));
	dim3 HiddenOutgrid(iDivUp(SamNum, BLOCK_SIZE), iDivUp((HiddenUnitNum), BLOCK_SIZE));
	dim3 SamOutgrid(iDivUp(SamNum, BLOCK_SIZE), iDivUp(OutDim, BLOCK_SIZE));

	dim3 OutErrGrid(iDivUp(OutDim, BLOCK_SIZE), iDivUp(OutDim, BLOCK_SIZE));

/*
	for(int i=0;i<SamNum;i++)
	{
		for(int j=0;j<InDim+1;j++)
		{
			printf("%.4f\t", h_SamInEx[IDX2C(i,j,SamNum)]);
		}
		printf("\n");
	}
*/

	float errOld;
	for(int i=0; i< OutDim; i++)
	{
		CUDA_SAFE_CALL(hipMemcpy(d_W2 + HiddenUnitNum * i, d_W2Ex + (HiddenUnitNum + 1) * i, sizeof(float)*HiddenUnitNum,hipMemcpyDeviceToDevice));
		//ע��������column-majoring
	}
	hipblasSgemm('n','n',SamNum, HiddenUnitNum, (InDim+1), 1.0f,  d_SamInEx, SamNum, d_W1Ex, (InDim+1), 0.0f, HiddenOutEx, SamNum );

	logsig1<<<HiddenOutExgrid, threads>>>(HiddenOutEx, SamNum, HiddenUnitNum);
	//logsig1��߽����ֵ��Ϊ1
	hipblasSgemm('n','n',SamNum, OutDim, (HiddenUnitNum+1), 1.0f,  HiddenOutEx, SamNum, d_W2Ex, (HiddenUnitNum+1), 0.0f, NetworkOut, SamNum );
	logsig2<<<SamOutgrid,threads>>>(NetworkOut, SamNum, OutDim);



	float* h_Out;
	h_Out = (float*) malloc(sizeof(float) * size_SamOut);
	CUDA_SAFE_CALL(hipMemcpy(h_Out, NetworkOut, sizeof(float)*size_SamOut, hipMemcpyDeviceToHost));

	int rightCount = 0;

	float door = (HIGH - LOW) * 0.7 + LOW;
	for(int i=0;i<SamNum;i++)
	{
		int predict = 0;
		for(int j=0;j<OutDim;j++)
		{
			predict <<= 1;
			float tar = h_Out[IDX2C(i,j, SamNum)];
			if(tar > door)
			{
				predict |= 1;
			}
			
			printf("%.3f\t", tar);
		}
		printf("\n");
		printf("%d -> %d\n", imageList[i].label, predict );
		if(imageList[i].label == predict)
		{
			rightCount++;
		}
	}

	printf("Percent = %.2f %%\n", (float)rightCount * 100 / SamNum);
	
	free(h_Out);

	//logsig2����߽���ֵ

	CUDA_SAFE_CALL(hipFree(d_SamInEx));
	CUDA_SAFE_CALL(hipFree(d_W1Ex));
	CUDA_SAFE_CALL(hipFree(d_W2Ex));
	CUDA_SAFE_CALL(hipFree(d_W2));

	CUDA_SAFE_CALL(hipFree(HiddenOutEx));
	CUDA_SAFE_CALL(hipFree(NetworkOut));

}

void Print(float* arr, int row, int col, const char* str)
{
	float* h_arr;

	h_arr = (float*) malloc(sizeof(float) * row * col);

	if(h_arr == NULL)
	{
		return;
	}

	CUDA_SAFE_CALL(hipMemcpy(h_arr, arr, row*col*sizeof(float), hipMemcpyDeviceToHost));

	PrintHost(h_arr, row, col, str);
	free(h_arr);
}

void PrintHost(float* arr, int row, int col, const char* str)
{
	if(str != NULL)
	{
		printf("%s\n", str);
	}

	for(int i=0;i<row;i++)
	{
		for(int j=0;j<col;j++)
		{
			printf("%f\t", arr[IDX2C(i,j,row)]);
		}
		printf("\n");
	}
}


