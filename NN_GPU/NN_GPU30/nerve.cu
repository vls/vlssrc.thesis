#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* BP nerve neworl research
 * Host code.
 */
#include "nerve.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil.h>
#include <hipblas.h>
#include <nerve_kernel.h>
#include "Image.h"
#include "Reduce.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

////////////////////////////////////////////////////////////////////////////////
// declaration, forward




////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
const int OUTPUT_NUM = 4;
int
run(int argc, char** argv)
{	
	const float lr = 0.1f;
	const int MaxEpochs = 1;
	const int HiddenUnitNum = 10;
	const int InDim = 2;
	const int OutDim = 3;
	const int SamNum = 200;

	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));

	float* h_SamInEx;
	float* h_SamOut;
	float* h_W1Ex;
	float* h_W2Ex;
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	CUT_DEVICE_INIT(argc, argv);
	printf("%s\n", hipGetErrorString(hipGetLastError()));

	
	hipblasStatus_t status;
	status = cublasInit();
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		printf("Can't init cublas\n");
		printf("%s\n", hipGetErrorString(hipGetLastError()));
		return -1;
	}
	
	printf("%s\n", hipGetErrorString(hipGetLastError()));
	

	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamInEx, sizeof(float)*size_SamInEx));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamOut, sizeof(float)*size_SamOut));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_W1Ex, sizeof(float)*size_W1Ex));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_W2Ex, sizeof(float)*size_W2Ex));
	
	if(!InitSample(SamNum, InDim, OutDim, HiddenUnitNum, h_SamInEx, h_SamOut, h_W1Ex, h_W2Ex))
	{
		printf("Can't init sample\n");
		return 0;
	}
    train(lr, MaxEpochs, HiddenUnitNum, InDim, OutDim, SamNum, h_SamInEx, h_SamOut, h_W1Ex, h_W2Ex);
	/*
	printf("\n");
	for(int i=0; i<size_W1Ex; i++)
	{
		printf("%3.3f ",h_W1Ex[i]);
	}

	printf("\n");
	for(int i=0; i<size_W2Ex; i++)
	{
		printf("%3.3f ",h_W2Ex[i]);
	}
	*/
	FILE *p;
	p = fopen("nervedata0_W1Ex.dat", "wb");
	fwrite(h_W1Ex,sizeof(float),size_W1Ex,p);
	fclose(p);

	p = fopen("nervedata0_W2Ex.dat", "wb");
	fwrite(h_W2Ex,sizeof(float),size_W2Ex,p);
	fclose(p);


	CUDA_SAFE_CALL( hipHostFree((h_SamInEx)));
	CUDA_SAFE_CALL( hipHostFree((h_SamOut)));
	CUDA_SAFE_CALL( hipHostFree((h_W1Ex)));
	CUDA_SAFE_CALL( hipHostFree((h_W2Ex)));

	h_SamInEx = NULL;
	h_SamOut = NULL;
	h_W1Ex=NULL;
	h_W2Ex=NULL;

	cublasShutdown();
	CUT_EXIT(argc, argv);

	printf("run complete\n");
}


int runImage(int argc, char** argv, Image* imageList, int count, int maxIter)
{
	if(imageList == NULL || count == 0)
	{
		return -1;
	}

	const float lr = 0.1f;
	const int MaxEpochs = maxIter;
	const int HiddenUnitNum = 16;
	const int InDim = imageList[0].length;
	const int OutDim = OUTPUT_NUM;
	const int SamNum = count;

	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));

	float* h_SamInEx;
	float* h_SamOut;
	float* h_W1Ex;
	float* h_W2Ex;
	CUT_DEVICE_INIT(argc, argv);


	hipblasStatus_t status;
	status = cublasInit();
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		printf("Can't init cublas\n");
		printf("%s\n", hipGetErrorString(hipGetLastError()));
		return -1;
	}


	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamInEx, sizeof(float)*size_SamInEx));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_SamOut, sizeof(float)*size_SamOut));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_W1Ex, sizeof(float)*size_W1Ex));
	CUDA_SAFE_CALL( hipHostMalloc((void**) &h_W2Ex, sizeof(float)*size_W2Ex));

	if(!InitImage(SamNum, InDim, OutDim, HiddenUnitNum, h_SamInEx, h_SamOut, h_W1Ex, h_W2Ex, imageList))
	{
		printf("Can't init image into input\n");
		return 0;
	}
	train(lr, MaxEpochs, HiddenUnitNum, InDim, OutDim, SamNum, h_SamInEx, h_SamOut, h_W1Ex, h_W2Ex);
	


	CUDA_SAFE_CALL( hipHostFree((h_SamInEx)));
	CUDA_SAFE_CALL( hipHostFree((h_SamOut)));
	CUDA_SAFE_CALL( hipHostFree((h_W1Ex)));
	CUDA_SAFE_CALL( hipHostFree((h_W2Ex)));

	h_SamInEx = NULL;
	h_SamOut = NULL;
	h_W1Ex=NULL;
	h_W2Ex=NULL;

	cublasShutdown();
	CUT_EXIT(argc, argv);

	printf("run complete\n");
}

int 
iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}


bool InitImage(int SamNum, int InDim, int OutDim, int HiddenUnitNum, float* h_SamInEx, float* h_SamOut, float* h_W1Ex,float* h_W2Ex, Image* imageList)
{
	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));
	const int TARGET_NUM = 10;
	

	for(int i=0; i<size_SamInEx; i++)
	{
		h_SamInEx[i] = 1.0f;
	}

	for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<InDim;j++)
		{
			h_SamInEx[IDX2C(i,j,SamNum)] = imageList[i].content[j] / 16;
		}
	}
/*
	#ifdef _DEBUG
	printf("Input:\n");
for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<InDim;j++)
		{
			printf("%.6f\t", h_SamInEx[IDX2C(i,j,SamNum)]);
		}
		printf("\n");
	}
#endif // _DEBUG
*/

	float target[TARGET_NUM][OUTPUT_NUM];
	for(int i=0;i<TARGET_NUM;i++)
	{
		int fi = i;
		int j = OUTPUT_NUM -1;
		memset(target[i], 0, sizeof(float) * OUTPUT_NUM);
		while(fi)
		{
			target[i][j--] =(float)( fi & 1 ? 1 : 0);
			fi >>= 1;
		}
	}

	for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<OutDim;j++)
		{
			
			h_SamOut[IDX2C(i,j,SamNum)] = target[imageList[i].label][j];
		}
	}
/*
	#ifdef _DEBUG
printf("Output:\n");
	for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<OutDim;j++)
		{

			printf("%.3f\t",h_SamOut[IDX2C(i,j,SamNum)]);
		}
		printf("\n");
	}
#endif // _DEBUG
*/
	for(int i=0; i<size_W1Ex; i++)
	{
		h_W1Ex[i]=0.2f*rand()/(float)RAND_MAX - 0.1f;
	}
	for(int i=0; i<size_W2Ex; i++)
	{
		h_W2Ex[i]=0.2f*rand()/(float)RAND_MAX - 0.1f;
	}

	return true;
}

bool InitSample(int SamNum, int InDim, int OutDim, int HiddenUnitNum, float* h_SamInEx, float* h_SamOut, float* h_W1Ex,float* h_W2Ex)
{
	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));


	srand(clock());
	FILE *p;
	p = fopen("nervedata\\samin.dat", "rb");

	if(!p) return false;

	for(int i=0; i<size_SamInEx; i++)
	{
		h_SamInEx[i] = 1.0f;
	}

	for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<InDim;j++)
		{
			fread(&h_SamInEx[j*SamNum+i],sizeof(float),1,p);
		}
	}
	fclose(p);

	for(int i=0; i<size_SamInEx; i++)
	{
		//printf("%3.4f\t",h_SamInEx[i]);
		//if((i + 1) % InDim == 0) printf("\n");
	}
	
	printf("\n");

	p = fopen("nervedata\\samout.dat", "rb");
	
	if(!p) return false;

	for(int i=0; i<SamNum; i++)
	{	
		for(int j=0; j<OutDim;j++)
		{
			fread(&h_SamOut[j*SamNum+i],sizeof(float),1,p);
		}
	}
	for(int i=0; i<size_SamOut; i++)
	{
		//printf("%3.5f\t",h_SamOut[i]);
		//if((i + 1) % OutDim == 0) printf("\n");
	}
	fclose(p);

	printf("\n");

	for(int i=0; i<size_W1Ex; i++)
	{
		h_W1Ex[i]=0.2f*rand()/(float)RAND_MAX - 0.1f;
	}
	for(int i=0; i<size_W2Ex; i++)
	{
		h_W2Ex[i]=0.2f*rand()/(float)RAND_MAX - 0.1f;
	}
/*
	p = fopen("I:\\temp\\NN_GPU\\nervedataW1Ex.dat", "rb");
	for(int i=0; i<(InDim+1); i++)
	{	
		for(int j=0; j<HiddenUnitNum;j++)
		{
			fread(&h_W1Ex[j*(InDim+1)+i],sizeof(float),1,p);
		}
	}

	for(int i=0; i<size_W1Ex; i++)
	{
		printf("%3.3f ",h_W1Ex[i]);
	}
	fclose(p);

	printf("\n");

	p = fopen("I:\\temp\\NN_GPU\\nervedataW2Ex.dat", "rb");
	for(int i=0; i<(HiddenUnitNum+1); i++)
	{	
		for(int j=0; j<OutDim;j++)
		{
			fread(&h_W2Ex[j*(HiddenUnitNum+1)+i],sizeof(float),1,p);
		}
	}

	for(int i=0; i<size_W2Ex; i++)
	{
		printf("%3.3f ",h_W2Ex[i]);
	}
	fclose(p);
*/
	printf("\n");

	p=NULL;

	return true;
}



void
train(float lr, int MaxEpochs, int HiddenUnitNum, int InDim,int OutDim,int SamNum, float* h_SamInEx, float* h_SamOut, float* h_W1Ex, float* h_W2Ex)
{
	const int size_SamInEx = ((int)(SamNum*(InDim+1)));
	const int size_SamOut =  ((int)(SamNum*OutDim));
	const int size_W1Ex	 = ((int)(HiddenUnitNum*(InDim+1)));
	const int size_W2Ex	 = ((int)(OutDim*(HiddenUnitNum+1)));

    float* d_W1Ex;
	float* d_W2Ex;
	float* d_W2;
	float* d_SamInEx;
	float* d_SamOut;
	float* d_error;
	float* HiddenOutEx;
	float* NetworkOut;
	float* Delta1;
	float* Delta2;

	unsigned int timer = 0;

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 HiddenOutExgrid(iDivUp(SamNum, BLOCK_SIZE), iDivUp((HiddenUnitNum+1), BLOCK_SIZE));
	dim3 HiddenOutgrid(iDivUp(SamNum, BLOCK_SIZE), iDivUp((HiddenUnitNum), BLOCK_SIZE));
	dim3 SamOutgrid(iDivUp(SamNum, BLOCK_SIZE), iDivUp(OutDim, BLOCK_SIZE));

	dim3 OutErrGrid(iDivUp(OutDim, BLOCK_SIZE), iDivUp(OutDim, BLOCK_SIZE));

	float* h_HiddenOutEx;
	h_HiddenOutEx = (float*) malloc(sizeof(float)*(HiddenUnitNum+1)*SamNum);

	CUDA_SAFE_CALL(hipMalloc((void**) &d_error, sizeof(float)*size_SamOut));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_W1Ex, sizeof(float)*size_W1Ex));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_W2Ex, sizeof(float)*size_W2Ex));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_W2, sizeof(float)*OutDim*HiddenUnitNum));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_SamInEx, sizeof(float)*size_SamInEx));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_SamOut, sizeof(float)*size_SamOut));
	CUDA_SAFE_CALL(hipMalloc((void**) &HiddenOutEx, sizeof(float)*(HiddenUnitNum+1)*SamNum));
	CUDA_SAFE_CALL(hipMalloc((void**) &NetworkOut, sizeof(float)*size_SamOut));
	CUDA_SAFE_CALL(hipMalloc((void**) &Delta1, sizeof(float)*HiddenUnitNum*SamNum));
	CUDA_SAFE_CALL(hipMalloc((void**) &Delta2, sizeof(float)*size_SamOut));  
    CUDA_SAFE_CALL(hipMemcpy(d_W1Ex, h_W1Ex, sizeof(float)*size_W1Ex,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_W2Ex, h_W2Ex, sizeof(float)*size_W2Ex,hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_SamInEx, h_SamInEx, sizeof(float)*size_SamInEx,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_SamOut, h_SamOut, sizeof(float)*size_SamOut,hipMemcpyHostToDevice));

	CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));
	float err;
	for(int l=0; l< MaxEpochs; l++)
	{

		for(int i=0; i< OutDim; i++)
		{
			CUDA_SAFE_CALL(hipMemcpy(d_W2 + HiddenUnitNum * i, d_W2Ex + (HiddenUnitNum + 1) * i, sizeof(float)*HiddenUnitNum,hipMemcpyDeviceToDevice));
			//ע��������column-majoring
		}
		
		hipblasSgemm('n','n',SamNum, HiddenUnitNum, (InDim+1), 1.0f,  d_SamInEx, SamNum, d_W1Ex, (InDim+1), 0.0f, HiddenOutEx, SamNum );
		//SamNum*(InDim+1) * (InDim+1)*HiddenUnitNUm
/*
		#ifdef _DEBUG
			CUDA_SAFE_CALL(hipMemcpy(h_HiddenOutEx, HiddenOutEx, sizeof(float)*(HiddenUnitNum+1)*SamNum, hipMemcpyDeviceToHost));
		#endif // _DEBUG
*/

		logsig1<<<HiddenOutExgrid, threads>>>(HiddenOutEx, SamNum, HiddenUnitNum);
		//logsig1��߽����ֵ��Ϊ1
		hipblasSgemm('n','n',SamNum, OutDim, (HiddenUnitNum+1), 1.0f,  HiddenOutEx, SamNum, d_W2Ex, (HiddenUnitNum+1), 0.0f, NetworkOut, SamNum );
		logsig2<<<SamOutgrid,threads>>>(NetworkOut, SamNum, OutDim);
		//logsig2����߽���ֵ
		dotsub<<<SamOutgrid,threads>>>(Delta2, d_SamOut, NetworkOut, SamNum, OutDim);
//#ifdef VERBOSE
		err = GPUPowSum(Delta2, OutDim* SamNum);
		//printf("Iter = %d, Err = %.6f\n", l, err);
//#endif //VERBOSE

		getdelta<<<SamOutgrid,threads>>>(Delta2, NetworkOut, SamNum, OutDim);
		hipblasSgemm('t','n',(HiddenUnitNum+1), OutDim, SamNum, lr,  HiddenOutEx, SamNum, Delta2, SamNum, 1.0f, d_W2Ex, (HiddenUnitNum+1) );
		//ʵ���Ͻ�W�仯����������
		hipblasSgemm('n','t', SamNum, HiddenUnitNum, OutDim, 1.0f,  Delta2, SamNum, d_W2, HiddenUnitNum, 0.0f, Delta1, SamNum );
		getdelta<<<HiddenOutgrid, threads>>>(Delta1, HiddenOutEx, SamNum, HiddenUnitNum);
		hipblasSgemm('t','n', (InDim+1), HiddenUnitNum, SamNum, lr, d_SamInEx, SamNum, Delta1, SamNum, 1.0f, d_W1Ex, (InDim+1));
	}
	CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL( cutDeleteTimer( timer));
	printf("Err = %.6f\n",err);

	CUDA_SAFE_CALL(hipMemcpy(h_W1Ex, d_W1Ex, sizeof(float)*size_W1Ex,hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(h_W2Ex, d_W2Ex, sizeof(float)*size_W2Ex,hipMemcpyDeviceToHost));
/*	
#ifdef _DEBUG
	for(int i=0;i<SamNum;i++)
	{
		for(int j = 0;j<HiddenUnitNum +1;j++)
		{
			printf("%.3f\t", h_HiddenOutEx[IDX2C(i,j,SamNum)]);
		}
		printf("\n");
	}
#endif // _DEBUG
	*/

    CUDA_SAFE_CALL(hipFree(d_SamInEx));
    CUDA_SAFE_CALL(hipFree(d_SamOut));
    CUDA_SAFE_CALL(hipFree(d_W1Ex));
	CUDA_SAFE_CALL(hipFree(d_W2Ex));
	
	CUDA_SAFE_CALL(hipFree(HiddenOutEx));
	CUDA_SAFE_CALL(hipFree(NetworkOut));
	CUDA_SAFE_CALL(hipFree(Delta1));
	CUDA_SAFE_CALL(hipFree(Delta2));

	free(h_HiddenOutEx);
}


