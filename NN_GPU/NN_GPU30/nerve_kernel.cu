/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* BP nerve neworl research
 * Device code.
 */

#ifndef _NERVE_KERNEL_H_
#define _NERVE_KERNEL_H_

#include <stdio.h>
// Matrix dimensions
// (chosen as multiples of the thread block size for simplicity)
#define BLOCK_SIZE 16

////////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: C = A * B
//! wA is A's width and wB is B's width
////////////////////////////////////////////////////////////////////////////////
__global__ void
logsig1( float* A, int wA, int hA)
{
    const int posy = blockIdx.y * blockDim.y + threadIdx.y;
	const int posx = blockIdx.x * blockDim.x + threadIdx.x;
	const int pos = posy * wA + posx;

	if(posx < wA )
	{
	if(posy < hA )
    A[pos] = 1/(1+expf(-A[pos]));
	else if(posy == hA)
	A[pos] = 1.0f;
	}
}

__global__ void
logsig2( float* A, int wA, int hA)
{
    const int posy = blockIdx.y * blockDim.y + threadIdx.y;
	const int posx = blockIdx.x * blockDim.x + threadIdx.x;
	const int pos = posy * wA + posx;

	if((posx < wA)&&(posy < hA))
	{
    A[pos] = 1/(1+expf(-A[pos]));
	}
}

__global__ void
dotsub( float* C, float* A, float* B, int wA, int hA)
{
    const int posy = blockIdx.y * blockDim.y + threadIdx.y;
	const int posx = blockIdx.x * blockDim.x + threadIdx.x;
	const int pos = posy * wA + posx;

	if((posx < wA)&&(posy < hA))
	{
    C[pos] = A[pos]-B[pos];
	}
}

__global__ void
getError(float* diffMat, int w, int h)
{

}

__global__ void
getdelta( float* C, float* A, int wA, int hA)
{
    //const int tidx = threadIdx.x;
	//const int tidy = threadIdx.y;
	const int posy = blockIdx.y * blockDim.y + threadIdx.y;
	const int posx = blockIdx.x * blockDim.x + threadIdx.x;
	const int pos = posy * wA + posx;

	//__shared__ s_A[BLOCK_SIZE][BLOCK_SIZE];

	if((posx < wA)&&(posy < hA))
	{
	//s_A[tidy][tidx] = A[pos];
    C[pos] *= ((1.0f - A[pos])*A[pos]);
	}
	__syncthreads();
}

#endif
