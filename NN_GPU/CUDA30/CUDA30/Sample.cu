#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>


#if __DEVICE_EMULATION__

bool InitCUDA(void){return true;}

#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);

	printf("CUDA initialized.\n");
	
	int driverVersion, runtimeVersion;
	hipDriverGetVersion(&driverVersion);
	printf("  CUDA Driver Version:                           %d.%d\n", driverVersion/1000, driverVersion%100);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("  CUDA Runtime Version:                          %d.%d\n", runtimeVersion/1000, runtimeVersion%100);
	return true;
}

#endif
